#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <chrono>
#include <math.h>
#include <fstream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// Maximum number of blocks in a device grid (for each dim)
#define MAX_BLOCKS 65535

// Min function
#define MIN(a,b) (((a)<(b))?(a):(b))

struct Pixel {
    uint8_t r;
    uint8_t g;
    uint8_t b;
};

__host__ void initialize_image(Pixel **image, const int width, const int height) {
    // Initialize image
    for (int i = 0; i < width * height; i++) {
        (*image)[i].r = 0;
        (*image)[i].g = 0;
        (*image)[i].b = 0;
    }
}

__host__ void create_image(Pixel **image, const string image_path, int *width, int *height, int *channels, const int byte_stride) {
    // Read image
    unsigned char *file = stbi_load(image_path.c_str(), width, height, channels, byte_stride);

    // Allocate memory for image
    *image = new Pixel[*width * *height];
    initialize_image(image, *width, *height);

    // Create image
    for (int i = 0; i < *width * *height; i++) {
        (*image)[i].r = (uint8_t) file[i * byte_stride];
        (*image)[i].g = (uint8_t) file[i * byte_stride + 1];
        (*image)[i].b = (uint8_t) file[i * byte_stride + 2];
    }

    // Free memory
    stbi_image_free(file);
}

__host__ void write_image(Pixel **out, const string output_path, const int width, const int height, const int channels, const int byte_stride) {
    // Allocate memory for output image
    unsigned char *file = new unsigned char[width * height * byte_stride];

    // Create output image
    for (int i = 0; i < width * height; i++) {
        file[i * byte_stride] = (*out)[i].r;
        file[i * byte_stride + 1] = (*out)[i].g;
        file[i * byte_stride + 2] = (*out)[i].b;
    }

    // Write output image
    stbi_write_png(output_path.c_str(), width, height, channels, file, width * byte_stride);

    // Free memory
    delete[] file;
}

__device__ uint16_t clamp(const double value) {
    if (value < 0) {
        return 0;
    } else if (value > 255) {
        return 255;
    } else {
        return (uint16_t) value;
    }
}

__device__ void frobenius_norm(Pixel **out, Pixel **in, const double *kernel, const int width, const int height, const int kernel_size, const int x, const int y) {
    // Image Shift
    const int shift = kernel_size / 2;
    int y_shift = 0;
    int x_shift = 0;

    // Cuda image, out and kernel index (host and device index are the same because we spawn one thread per pixel)
    int img_index = 0;
    int kernel_index = 0;
    const int out_index = y * width + x;

    // Output
    double out_r = 0.0;
    double out_g = 0.0;
    double out_b = 0.0;

    // Compute frobenius norm
    for (int j = 0; j < kernel_size; j++) {
        // Compute shift in y direction
        y_shift = y + j - shift;
        for (int i = 0; i < kernel_size; i++) {
            // Compute shift in x direction
            x_shift = x + i - shift;

            // Check if pixel is in image (if not, skip i.e. use 0 padding)
            if (x_shift < 0 || x_shift > width - 1 || y_shift < 0 || y_shift > height - 1) {
                continue;
            }

            // Compute index
            img_index = y_shift * width + x_shift;
            kernel_index = j * kernel_size + i;
            

            // Compute output
            out_r += (double) (*in)[img_index].r * kernel[kernel_index];
            out_g += (double) (*in)[img_index].g * kernel[kernel_index];
            out_b += (double) (*in)[img_index].b * kernel[kernel_index];
        }
    }

    // Clamp output
    (*out)[out_index].r = clamp(out_r);
    (*out)[out_index].g = clamp(out_g);
    (*out)[out_index].b = clamp(out_b);
}

__global__ void conv2D(Pixel **out, Pixel **in, const double *kernel, const int width, const int height, const int kernel_size) {
    // Compute convolution for each pixel
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    frobenius_norm(out, in, kernel, width, height, kernel_size, x, y);
}

__host__ const double* determine_kernel(const string kernel_choice) {
    const double *kernel;
    if (kernel_choice == "blur") {
        kernel = new const double[9] {
            1/9.0, 1/9.0, 1/9.0,
            1/9.0, 1/9.0, 1/9.0,
            1/9.0, 1/9.0, 1/9.0
        };
    } else if (kernel_choice == "sharpen") {
        kernel = new const double[9] {
            0, -1, 0,
            -1, 5, -1,
            0, -1, 0
        };
    } else if (kernel_choice == "edge") {
        kernel = new const double[9] {
            -1, -1, -1,
            -1, 8, -1,
            -1, -1, -1
        };
    } else if (kernel_choice == "emboss") {
        kernel = new const double[9] {
            -2, -1, 0,
            -1, 1, 1,
            0, 1, 2
        };
    } else {
        kernel = new const double[9] {
            1.0, 1.0, 1.0,
            1.0, 1.0, 1.0,
            1.0, 1.0, 1.0
        };
    }
    return kernel;
}

__host__ void process_image(const string image_path, const string kernel_choice, const string output_path, const int nthreads_per_block) {
    // Create image on host
    int width, height, channels;
    const int byte_stride = 3;
    Pixel *image;
    create_image(&image, image_path, &width, &height, &channels, byte_stride);

    // Create image on device
    int width_d, height_d, channels_d;
    const int byte_stride_d;
    Pixel *image_d;
    hipMalloc(&width_d, sizeof(int));
    hipMalloc(&height_d, sizeof(int));
    hipMalloc(&channels_d, sizeof(int));
    hipMalloc(&byte_stride_d, sizeof(int));
    hipMalloc(image_d, width * height * byte_stride * sizeof(Pixel));

    // Copy image to device
    hipMemcpy(&width_d, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&height_d, &height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&channels_d, &channels, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&byte_stride_d, &byte_stride, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(image_d, image, width * height * byte_stride * sizeof(Pixel), hipMemcpyHostToDevice);

    // Determine kernel on host
    const int kernel_size = 3;
    const double *kernel = determine_kernel(kernel_choice);

    // Create kernel on device
    const int kernel_size_d;
    const double *kernel_d;
    hipMalloc(&kernel_size_d, sizeof(int));
    hipMalloc(kernel_d, kernel_size * kernel_size * sizeof(double));

    // Copy kernel to device
    hipMemcpy(&kernel_size_d, &kernel_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_d, kernel, kernel_size * kernel_size * sizeof(double), hipMemcpyHostToDevice);

    // Allocate memory for output image on host and initialize
    Pixel *out = new Pixel[width * height];
    initialize_image(&out, width, height);

    // Allocate memory for output image on device
    Pixel *out_d;
    hipMalloc(out_d, width * height * sizeof(Pixel));
    hipMemcpy(out_d, out, width * height * sizeof(Pixel), hipMemcpyHostToDevice);

    // Compute the number of blocks
    const int x_blocks = MIN((width/n_threads_per_block) + 1, MAX_BLOCKS);
    const int y_blocks = MIN((height/n_threads_per_block) + 1, MAX_BLOCKS);
    const dim3 block_size(n_threads_per_block, n_threads_per_block);
    const dim3 grid_size(x_blocks, y_blocks);

    // CUDA timer
    hipEvent_t start_device, stop_device;  
    float time_device;

    // Create timers
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);

    // Start timer
    hipEventRecord(start_device, 0);  

    // Compute convolution
    conv2D<<<grid_size, block_size>>>(out_d, image_d, kernel_d, width_d, height_d, kernel_size_d);

    // Stop timer
    hipEventRecord(stop_device, 0);
    hipEventSynchronize(stop_device);
    hipEventElapsedTime(&time_device, start_device, stop_device);

    // Compute time
    cout << "Time: " << time_device << " ms" << endl;

    // Copy output image from device to host
    hipMemcpy(out, out_d, width * height * sizeof(Pixel), hipMemcpyDeviceToHost);

    // Write output image
    write_image(&out, output_path, width, height, channels, byte_stride);
    

    // Free memory on host
    delete[] image;
    delete[] out;
    delete[] kernel;

    // Free memory on device
    hipFree(&width_d);
    hipFree(&height_d);
    hipFree(&channels_d);
    hipFree(&byte_stride_d);
    hipFree(&kernel_size_d);
    hipFree(image_d);
    hipFree(out_d);
    hipFree(kernel_d);

}

__host__ int main(int argc, char** argv) {
    // Get the input args
    const string image_path = argv[1];
    const string kernel_choice = argv[2];
    const string output_path = argv[3];
    const int nthreads_per_block = atoi(argv[4]);

    // Process image
    process_image(image_path, kernel_choice, output_path, nthreads_per_block);

    return 0;
}